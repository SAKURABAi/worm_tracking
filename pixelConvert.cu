#include "hip/hip_runtime.h"

#ifndef _PIXEL_CONVERT_H_
#define _PIXEL_CONVERT_H_

#include "Util.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <>

__global__  void rgb8_kernel(uchar3* dev_output_data, uchar* dev_original_data, int width, int height)
{
	int col = blockIdx.x*  blockDim.x + threadIdx.x;
	int row = blockIdx.y*  blockDim.y + threadIdx.y;
	if (col<width && row<height){
		dev_output_data[row*width + col].x = dev_original_data[row*width + col];
		dev_output_data[row*width + col].y = dev_original_data[row*width + col];
		dev_output_data[row*width + col].z = dev_original_data[row*width + col];
	}
}

//pixel convert for 8-bits
/*Parameteters:
dev_output_data:  output image (RGB) in the device memory;
dev_original_data: original image in the device memory;
width:   image width;
height: image height
*/
extern "C"
int pixelConvert8(uchar3* dev_output_data, uchar* dev_original_data, int width, int height)
{
	dim3 Db = dim3(32, 32);
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);
	rgb8_kernel << <Dg, Db >> >(dev_output_data, dev_original_data, width, height);
	hipDeviceSynchronize();

	hipError_t error = hipGetLastError();
	if (error != hipSuccess){
		printf("rgb8_kernel() failed to launch, error = %d\n", error);
		return _CUDA_LAUNCH_FAILURE;
	}
	return _CUDA_LAUNCH_SUCCESS;
}

#endif //_PIXEL_CONVERT_H_